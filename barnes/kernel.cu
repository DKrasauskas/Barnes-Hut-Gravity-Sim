﻿
#include "hip/hip_runtime.h"
#include ""
#include <thrust/execution_policy.h>
#include <stdio.h>
#include <corecrt_malloc.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/gather.h>
#include <thrust/iterator/zip_iterator.h>
#define uint unsigned int

//gpu:
#include "include/glad/glad.h"
#include "include/GLFW/glfw3.h"
#include "Shader.h"
#include "Buffer.h"
#include "grid.h"
int p = 2;
#include "settings.h"
#include "math.h"
#include "zOrder.h"
#include "points.h"

//_________________________________________________________________________________________OpenGL_____________________________________________________________________________//

void framebuffer_size_callback(GLFWwindow* window, int width, int height);
void processInput(GLFWwindow* window);

GLFWwindow* createWindow() {
    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 6);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
    GLFWwindow* win = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "Orbits", NULL, NULL);
    win == NULL ? throw std::exception() : NULL;
    glfwMakeContextCurrent(win);
    !gladLoadGLLoader((GLADloadproc)glfwGetProcAddress) ? throw std::exception() : NULL;
    return win;
}
//_________________________________________________________________________________________Cuda_____________________________________________________________________________//
__managed__ float* ax, * bx, * cx, * dx, * scales, * px, * color;
#include "tree.h"
#include "timer.h"
#include "kernels.h"

int main()
{
    GLFWwindow* window = createWindow();
    glfwMakeContextCurrent(window);
    glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_HIDDEN);
    glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
    glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_HIDDEN);
    Shader vertex("vertex.glsl", "fragment.glsl");
    Shader vertex2("vertex1.glsl", "fragment.glsl");
    Cshader computes("compute.glsl");
    Grid gd = grid(160000, 1);
    Grid pt = grid(1000); //100;
    Buffer buff((void*)gd.vertices, (void*)gd.indices, gd.v_size, gd.i_size);
    Buffer buff2((void*)pt.vertices, (void*)pt.indices, pt.v_size, pt.i_size);
   
    //tree memaloc
    quad_tree tree(D_MAX);
  
    int n = 2;
    cin >> n;
    int size;
    int p_count = n;
    points pts = generate_random_points(n, N_DIM);
    hipMallocManaged(&ax, sizeof(float) * n);
    hipMallocManaged(&bx, sizeof(float) * n);
    hipMallocManaged(&color, sizeof(float) * n);
    hipMallocManaged(&cx, sizeof(float) * 1000 * 1024);
    hipMallocManaged(&dx, sizeof(float) * 1000 * 1024);
    hipMallocManaged(&scales, sizeof(float) * 1000 * 1024);
    GLuint buf1, buf2, buf3, buf4, buf5, col;
    glGenBuffers(1, &buf1);
    glGenBuffers(1, &buf2);
    glGenBuffers(1, &buf3);
    glGenBuffers(1, &buf4);
    glGenBuffers(1, &buf5);
    glGenBuffers(1, &col);
    tree.pt = pts;
    const char* cmd = "ffmpeg -r 60 -f rawvideo -pix_fmt rgba -s 1200x1200 -i - -threads 0 -preset fast -y -pix_fmt yuv420p -crf 32 -vf vflip output.mp4";
    unsigned char* image = (unsigned char*)malloc(sizeof(unsigned char) * SCR_WIDTH * SCR_HEIGHT * 4);
    // open pipe to ffmpeg's stdin in binary write mode
    FILE* ffmpeg = _popen(cmd, "wb");
    
    //render loop
    while (!glfwWindowShouldClose(window)) { 
        if (stateS > 2) {
            p_count = thrust::remove_if(tree.pt.data.begin(), tree.pt.data.begin() + p_count, boundary()) - tree.pt.data.begin();
            size = generate_bounding_boxes(tree, D_MAX, p_count);
            remove_nonleafs(tree, size);
            populate(tree, size);         
            traverse_tree << <n / 200, 200 >> > (thrust::raw_pointer_cast(&tree.ids[0]), thrust::raw_pointer_cast(&tree.clen[0]), thrust::raw_pointer_cast(&tree.nlen[0]), thrust::raw_pointer_cast(&tree.pt.data[0]), thrust::raw_pointer_cast(&tree.bdata[0]), thrust::raw_pointer_cast(&tree.keys[0]), size, p_count);
            hipDeviceSynchronize();          
            cpy_kernel << <n / 200, 200 >> > (ax, bx, color, thrust::raw_pointer_cast(&tree.pt.data[0]), thrust::raw_pointer_cast(&tree.pt.keys[0]), p_count);
            hipDeviceSynchronize();         
            if (OUT) {
                glReadPixels(0, 0, SCR_WIDTH, SCR_HEIGHT, GL_RGBA, GL_UNSIGNED_BYTE, image);
                fwrite(image, SCR_WIDTH * SCR_HEIGHT * sizeof(int), 1, ffmpeg);
            }               
        }  
        processInput(window);
        glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);
        if (stateS > 100) {
            boxes << <size / 100, 100 >> > (thrust::raw_pointer_cast(&tree.keys[0]), cx, dx, scales);
            hipDeviceSynchronize();
            glUseProgram(vertex.ID);
            glBindBuffer(GL_SHADER_STORAGE_BUFFER, buf3);
            glBufferData(GL_SHADER_STORAGE_BUFFER, sizeof(float) * size, cx, GL_STATIC_DRAW); 
            glBindBufferBase(GL_SHADER_STORAGE_BUFFER, 3, buf3);
            glBindBuffer(GL_SHADER_STORAGE_BUFFER, buf4);
            glBufferData(GL_SHADER_STORAGE_BUFFER, sizeof(float) * size, dx, GL_STATIC_DRAW); 
            glBindBufferBase(GL_SHADER_STORAGE_BUFFER, 4, buf4);
            glBindBuffer(GL_SHADER_STORAGE_BUFFER, buf5);
            glBufferData(GL_SHADER_STORAGE_BUFFER, sizeof(float) * size, scales, GL_STATIC_DRAW);
            glBindBufferBase(GL_SHADER_STORAGE_BUFFER, 5, buf5);
            glBindVertexArray(buff.VAO);
            glDrawElements(GL_LINES, 160000, GL_UNSIGNED_INT, (void*)0);
        }     
        glUseProgram(vertex2.ID);   
        glBindBuffer(GL_SHADER_STORAGE_BUFFER, buf1);
        glBufferData(GL_SHADER_STORAGE_BUFFER, sizeof(float) * (p_count) ,ax , GL_STATIC_DRAW); 
        glBindBufferBase(GL_SHADER_STORAGE_BUFFER, 1, buf1);
        glBindBuffer(GL_SHADER_STORAGE_BUFFER, buf2);
        glBufferData(GL_SHADER_STORAGE_BUFFER, sizeof(float) * (p_count), bx , GL_STATIC_DRAW); 
        glBindBufferBase(GL_SHADER_STORAGE_BUFFER, 2, buf2);
        glBindBuffer(GL_SHADER_STORAGE_BUFFER, col);
        glBufferData(GL_SHADER_STORAGE_BUFFER, sizeof(float) * (p_count), color, GL_STATIC_DRAW); 
        glBindBufferBase(GL_SHADER_STORAGE_BUFFER, 3, col);
        glBindVertexArray(buff2.VAO);
        glDrawArrays(GL_POINTS, 0, p_count);
        glfwSwapBuffers(window);
        glfwPollEvents();
    }
    _pclose(ffmpeg);
    hipFree(ax);
    hipFree(bx);
    hipFree(cx);
    hipFree(dx);
    hipFree(scales);
    free(image);
    return 0;
}

void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    glViewport(0, 0, width, height);
}
